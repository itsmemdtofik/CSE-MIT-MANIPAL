#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define N 1024

/* A CUDA program which takes a string as input and determine the number of occurences 
of a character 'a' in the string.This program uses atomicAdd() function.*/


__global__ void CUDACount(char* A, int* Count){
    int idx = threadIdx.x;

    if(A[idx] == 'a')
        atomicAdd(Count, 1);
}

int main(){
    char A[N];
    char *d_A;

    int count=0;
    int *d_count;
    int res;

    hipError_t err = hipGetLastError();

    printf("Enter String : ");
    scanf("%s", A);
    printf("String : %s\n", A);

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 1 : %s\n", hipGetErrorString(err));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 2 : %s\n", hipGetErrorString(err));

    hipMalloc((void**)&d_A, strlen(A)*sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));
    hipMalloc((void**)&res, sizeof(int));

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 3 : %s\n", hipGetErrorString(err));

    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
    
    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 3.5 : %s\n", hipGetErrorString(err));
    
    hipMemcpy(d_A, A, strlen(A)*sizeof(char), hipMemcpyHostToDevice);


    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 4 : %s\n", hipGetErrorString(err));

    CUDACount<<<1, strlen(A)>>>(d_A, d_count);

    err = hipGetLastError();
    if(err != hipSuccess)
        printf("CUDA Error Occured 5 : %s\n", hipGetErrorString(err));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    

    hipMemcpy(&res, d_count, sizeof(int), hipMemcpyDeviceToHost);
    printf("Total Occurances of 'a' = %d\n", res);

    printf("Time taken: %d\n", elapsedTime);

    hipFree(d_A);
    hipFree(d_count);
    return 0;

}