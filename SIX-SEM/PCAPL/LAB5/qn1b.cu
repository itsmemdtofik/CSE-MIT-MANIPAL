#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

/**
*Write a program in cuda to add two vectors of length N Using
(a)N threads
*/
__global__ void vecAddKernel1b(int *A, int *B, int *C)
{
    int id = threadIdx.x;
    C[id] = A[id] + B[id];
}
int main()
{
    // host copies of variables A, B & C
    int N = 5;
    int A[N] = {1, 2, 3, 4, 5};
    int B[N] = {6, 7, 8, 9, 10};
    int C[N] = {0, 0, 0, 0, 0};
    // device copies of variables A, B & C
    int *d_A, *d_B, *d_C;
    // Allocate space for device copies of A, B, C
    int size = N * sizeof(int);
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    // Copy inputs to device
    hipMemcpy(d_A, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, &B, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    vecAddKernel1b<<<1, N>>>(d_A, d_B, d_C);
    // Copy result back to host
    hipError_t err = hipMemcpy(&C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
    }
    printf("1b. Vector Addition using N= %d threads within a block\n", N);
    printf("Elements of Array A are: ");
    for(int k=0;k<N;k++){
        printf("%d ", A[k]);
    }
    printf("\n");
    printf("Elements of Array B are: ");
    for(int k=0;k<N;k++){
        printf("%d ", B[k]);
    }
    printf("\n");
    printf("Resultant Array C is:");
    for(int k=0;k<N;k++){
        printf("%d ", C[k]);
    }
    printf("\n");
    printf("\n");
    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}