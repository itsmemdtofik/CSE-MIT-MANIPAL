#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

__global__ void parallelSort(int *a, int *b, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = 0;
    int data = a[i];
    if (i < N)
    {
        for (int j = 0; j < N; j++)
            if (a[j] < data || (a[j] == data && j < i))
                pos++;
            b[pos] = data;
        }
    }
    int main()
    {
    // host copies of variables A, B
        int N = 5;
        int A[N] = {2, 5, 4, 3, 1};
        int B[N];
    // device copies of variables A, B
        int *d_A, *d_B;
    // Allocate space for device copies of A, B
        int size = N * sizeof(int);
        hipMalloc((void **)&d_A, size);
        hipMalloc((void **)&d_B, size);
    // Copy inputs to device
        hipMemcpy(d_A, &A, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
        parallelSort<<<ceil(N / 2.0), 2>>>(d_A, d_B, N);
    // Copy result back to host
        hipError_t err = hipMemcpy(&B, d_B, size, hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
        }
        printf("2. Parallel Selection Sort\n");
        printf("Initial array A elements:\n");
        for (int k = 0; k < N; k++)
        {
            printf("%d ", A[k]);
        }
        printf("\nResultant sorted array B elements:\n");
        for (int k = 0; k < N; k++)
        {
            printf("%d ", B[k]);
        }
    // Cleanup
        hipFree(d_A);
        hipFree(d_B);
        return 0;
    }