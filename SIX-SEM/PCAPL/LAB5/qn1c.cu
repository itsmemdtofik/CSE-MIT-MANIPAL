#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

/*Implement a CUDA program to add two vectors of Length N by keeping
the number of threads per block as 256(constant) and vary the
the number of blocks*/

__global__ void vecAddKernel(int *A, int *B, int *C, int N)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < N)
        C[id] = A[id] + B[id];
}
int main()
{
    // host copies of variables A, B & C
    int N = 5;
    int A[N] = {1, 2, 3, 4, 5};
    int B[N] = {6, 7, 8, 9, 10};
    int C[N] = {0, 0, 0, 0, 0};
    // device copies of variables A, B & C
    int *d_A, *d_B, *d_C;
    // Allocate space for device copies of A, B, C
    int size = N * sizeof(int);
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    // Copy inputs to device
    hipMemcpy(d_A, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, &B, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    vecAddKernel<<<ceil(N / 256.0), 256>>>(d_A, d_B, d_C, N);
    // Copy result back to host
    hipError_t err = hipMemcpy(&C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
    }
    printf("1c. Vector Addition using 256 threads per block and vary the number of blocks\n");
    printf("Elements of Array A are: ");
    for(int k=0;k<N;k++){
        printf("%d ", A[k]);
    }
    printf("\n");
    printf("Elements of Array B are: ");
    for(int k=0;k<N;k++){
        printf("%d ", B[k]);
    }
    printf("\n");
    printf("Resultant Array C is:");
    for(int k=0;k<N;k++){
        printf("%d ", C[k]);
    }
    printf("\n");
    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}