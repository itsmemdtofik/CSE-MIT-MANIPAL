#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

__global__ void oddEven(int *a, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        if (i % 2 == 1 && i < n - 1)
        {
            if (a[i] >= a[i + 1])
            {
                int t = a[i];
                a[i] = a[i + 1];
                a[i + 1] = t;
            }
        }
    }
}
__global__ void evenOdd(int *a, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        if (i % 2 == 0 && i < n - 1)
        {
            if (a[i] >= a[i + 1])
            {
                int t = a[i];
                a[i] = a[i + 1];
                a[i + 1] = t;
            }
        }
    }
}
int main()
{
    int N = 5;
    // host copies of variables A, B
    int A[N] = {2, 5, 4, 3, 1};
    printf("3. Sort using Odd Even Transposition Sorting:\n");
    printf("Initial Array A elements:\n");
    for (int k = 0; k < N; k++)
    {
        printf("%d ", A[k]);
    }
    int *d_A;
    int size = N * sizeof(int);
    hipMalloc((void **)&d_A, size);
    hipMemcpy(d_A, &A, size, hipMemcpyHostToDevice);
    for (int i = 0; i < N / 2; i++)
    {
        oddEven<<<ceil(N / 2.0), 2>>>(d_A, N);
        evenOdd<<<ceil(N / 2.0), 2>>>(d_A, N);
    }
    hipMemcpy(&A, d_A, size, hipMemcpyDeviceToHost);
    printf("\nResultant sorted Array A:\n");
    for (int i = 0; i < N; i++)
        printf("%d ", A[i]);
    return 0;
}